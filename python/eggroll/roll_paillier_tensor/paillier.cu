#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <sys/time.h>
#include "cgbn/cgbn.h"
#include "samples/utility/gpu_support.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// #include <chrono>
#include <cassert>

#include "fixedpoint.h"

// IMPORTANT:  DO NOT DEFINE TPI OR BITS BEFORE INCLUDING CGBN
#define TPI 32
#define CPH_BITS 2048 // cipher bits
#define MAX_RAND_SEED 4294967295U
#define WINDOW_BITS 5

// helpful typedefs for the kernel
typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, CPH_BITS> env_cph_t;
typedef cgbn_mem_t<CPH_BITS> gpu_cph;

void store2dev(void *address,  mpz_t z, unsigned int BITS) {
  size_t words;
  if(mpz_sizeinbase(z, 2)>BITS) {
    exit(1);
  }
  mpz_export((uint32_t *)address, &words, -1, sizeof(uint32_t), 0, 0, z);
  while(words<(BITS+31)/32)
    ((uint32_t *)address)[words++]=0;
}

void store2gmp(mpz_t z, void *address, unsigned int BITS ) {
  mpz_import(z, (BITS+31)/32, -1, sizeof(uint32_t), 0, 0, (uint32_t *)address);
}


void invert(mpz_t rop, mpz_t a, mpz_t b) {
  mpz_invert(rop, a, b);
}

inline void cudaMallocAndSet(void **addr, size_t size) {
  hipMalloc(addr, size);
  hipMemset(*addr, 0, size);
}

inline void dumpMem(char *addr, size_t size) {
  printf("dumping memory at 0x%x\n", addr);
  printf("0x");
  for (int64_t i = size - 1; i >= 0; i--) printf("%02x", addr[i] & 0xff);
  printf("\n");
}

enum MemcpyType {
HostToHost = 0,
HostToDevice,
DeviceToHost,
DeviceToDevice
};



class PaillierPublicKey {
 public:
  cgbn_mem_t<CPH_BITS> g;
  cgbn_mem_t<CPH_BITS> n;
  cgbn_mem_t<CPH_BITS> nsquare;
  cgbn_mem_t<CPH_BITS> max_int;
};


class PaillierPrivateKey {
 public:
  cgbn_mem_t<CPH_BITS> p;
  cgbn_mem_t<CPH_BITS> q;
  cgbn_mem_t<CPH_BITS> psquare;
  cgbn_mem_t<CPH_BITS> qsquare;
  cgbn_mem_t<CPH_BITS> q_inverse;
  cgbn_mem_t<CPH_BITS> hp;
  cgbn_mem_t<CPH_BITS> hq;
};

struct PaillierEncryptedNumber {
  char cipher[CPH_BITS/8]; // expected size: CPH_BITS/8 bytes
  int32_t exponent;
  int32_t base;
};

inline void extractPen(gpu_cph *dst, PaillierEncryptedNumber *src, uint32_t count, MemcpyType type) {
  for (int i = 0; i < count; i++) {
    if (type == HostToHost)
      memcpy(dst + i, src[i].cipher, sizeof(gpu_cph));
    else if (type == HostToDevice)
      hipMemcpy(dst + i, src[i].cipher, sizeof(gpu_cph), hipMemcpyHostToDevice);
    else if (type == DeviceToHost)
      hipMemcpy(dst + i, src[i].cipher, sizeof(gpu_cph), hipMemcpyDeviceToHost);
    else if (type == DeviceToDevice)
      hipMemcpy(dst + i, src[i].cipher, sizeof(gpu_cph), hipMemcpyDeviceToDevice);
  }
}

inline void penFromBuffer(PaillierEncryptedNumber *dst, gpu_cph *src, uint32_t count, MemcpyType type) {
  for (int i = 0; i < count; i++) {
    if (type == HostToHost)
      memcpy((dst + i)->cipher, src + i, sizeof(gpu_cph));
    else if (type == HostToDevice)
      hipMemcpy((dst + i)->cipher, src + i, sizeof(gpu_cph), hipMemcpyHostToDevice);
    else if (type == DeviceToHost)
      hipMemcpy((dst + i)->cipher, src + i, sizeof(gpu_cph), hipMemcpyDeviceToHost);
    else if (type == DeviceToDevice)
      hipMemcpy((dst + i)->cipher, src + i, sizeof(gpu_cph), hipMemcpyDeviceToDevice);
  }
}

// template<unsigned int _BITS, unsigned int _TPI>
__device__ __forceinline__ 
void mont_modular_power(env_cph_t &bn_env, env_cph_t::cgbn_t &result, 
		const env_cph_t::cgbn_t &x, const env_cph_t::cgbn_t &power, 
		const env_cph_t::cgbn_t &modulus) {
/************************************************************************************
* calculate x^power mod modulus with montgomery multiplication.
* input: x, power, modulus.
* output: result
* requirement: x < modulus and modulus is an odd number.
*/

  env_cph_t::cgbn_t         t, starts;
  int32_t      index, position, leading;
  uint32_t     mont_inv;
  env_cph_t::cgbn_local_t   odd_powers[1<<WINDOW_BITS-1];

  // find the leading one in the power
  leading=CPH_BITS-1-cgbn_clz(bn_env, power);
  if(leading>=0) {
    // convert x into Montgomery space, store in the odd powers table
    mont_inv=cgbn_bn2mont(bn_env, result, x, modulus);
    
    // compute t=x^2 mod modulus
    cgbn_mont_sqr(bn_env, t, result, modulus, mont_inv);
    
    // compute odd powers window table: x^1, x^3, x^5, ...
    cgbn_store(bn_env, odd_powers, result);
    #pragma nounroll
    for(index=1;index<(1<<WINDOW_BITS-1);index++) {
      cgbn_mont_mul(bn_env, result, result, t, modulus, mont_inv);
      cgbn_store(bn_env, odd_powers+index, result);
    }

    // starts contains an array of bits indicating the start of a window
    cgbn_set_ui32(bn_env, starts, 0);

    // organize p as a sequence of odd window indexes
    position=0;
    while(true) {
      if(cgbn_extract_bits_ui32(bn_env, power, position, 1)==0)
        position++;
      else {
        cgbn_insert_bits_ui32(bn_env, starts, starts, position, 1, 1);
        if(position+WINDOW_BITS>leading)
          break;
        position=position+WINDOW_BITS;
      }
    }

    // load first window.  Note, since the window index must be odd, we have to
    // divide it by two before indexing the window table.  Instead, we just don't
    // load the index LSB from power
    index=cgbn_extract_bits_ui32(bn_env, power, position+1, WINDOW_BITS-1);
    cgbn_load(bn_env, result, odd_powers+index);
    position--;
    
    // Process remaining windows 
    while(position>=0) {
      cgbn_mont_sqr(bn_env, result, result, modulus, mont_inv);
      if(cgbn_extract_bits_ui32(bn_env, starts, position, 1)==1) {
        // found a window, load the index
        index=cgbn_extract_bits_ui32(bn_env, power, position+1, WINDOW_BITS-1);
        cgbn_load(bn_env, t, odd_powers+index);
        cgbn_mont_mul(bn_env, result, result, t, modulus, mont_inv);
      }
      position--;
    }
    
    // convert result from Montgomery space
    cgbn_mont2bn(bn_env, result, result, modulus, mont_inv);
  }
  else {
    // p=0, thus x^p mod modulus=1
    cgbn_set_ui32(bn_env, result, 1);
  }
}

__device__  __forceinline__ void l_func(env_cph_t &bn_env, env_cph_t::cgbn_t &out, 
		env_cph_t::cgbn_t &cipher_t, env_cph_t::cgbn_t &x_t, env_cph_t::cgbn_t &xsquare_t, 
		env_cph_t::cgbn_t &hx_t) {
/****************************************************************************************
* calculate L(cipher_t^(x_t - 1) mod xsquare_t) * hx_t. 
* input: cipher_t, x_t, xsquare-t, hx_t
* out:   out
*/
  env_cph_t::cgbn_t  tmp, tmp2, cipher_lt;
  env_cph_t::cgbn_wide_t  tmp_wide;
  cgbn_sub_ui32(bn_env, tmp2, x_t, 1);
  
  if(cgbn_compare(bn_env, cipher_t, xsquare_t) >= 0) {
    cgbn_rem(bn_env, cipher_lt, cipher_t, xsquare_t);
    mont_modular_power(bn_env,tmp,cipher_lt,tmp2,xsquare_t);
  } else {
    mont_modular_power(bn_env, tmp, cipher_t, tmp2, xsquare_t);
  }
 
  cgbn_sub_ui32(bn_env, tmp, tmp, 1);
  cgbn_div(bn_env, tmp, tmp, x_t);
  cgbn_mul_wide(bn_env, tmp_wide, tmp, hx_t);
  cgbn_rem_wide(bn_env, tmp, tmp_wide, x_t);
 
  cgbn_set(bn_env, out, tmp);
}


__global__ void setup_kernel(hiprandState *state){
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ __noinline__ void apply_obfuscator(PaillierPublicKey *gpu_pub_key, cgbn_error_report_t *report, 
		gpu_cph *ciphers, gpu_cph *obfuscators, int count, hiprandState *state ) {
/******************************************************************************************
* obfuscate the encrypted text, obfuscator = cipher * r^n mod n^2
* in:
*   ciphers: encrypted text from simple raw encryption
*   state:   GPU random generator state.
* out:
*   obfuscators: obfused encryption text.
*/
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int tid= idx/TPI;
  if(tid>=count)
    return;

  context_t      bn_context(cgbn_report_monitor, report, tid);  
  env_cph_t     bn_env(bn_context.env<env_cph_t>());                   
  env_cph_t::cgbn_t  n, nsquare,cipher, r, tmp;
  env_cph_t::cgbn_wide_t tmp_wide;

  hiprandState localState = state[idx];
  unsigned int rand_r = hiprand_uniform(&localState) * MAX_RAND_SEED;                  
  state[idx] = localState;

  cgbn_set_ui32(bn_env, r, rand_r); // TODO: new rand or reuse
  cgbn_load(bn_env, n, &gpu_pub_key[0].n);
  cgbn_load(bn_env, nsquare, &gpu_pub_key[0].nsquare);
  cgbn_load(bn_env, cipher, &ciphers[tid]);
  mont_modular_power(bn_env, tmp, r, n, nsquare);
  cgbn_mul_wide(bn_env, tmp_wide, cipher, tmp);
  cgbn_rem_wide(bn_env, r, tmp_wide, nsquare);
  cgbn_store(bn_env, obfuscators + tid, r);   // store r into sum
}


__global__ void raw_encrypt(PaillierPublicKey *gpu_pub_key, cgbn_error_report_t *report, 
		gpu_cph *plains, gpu_cph *ciphers,int count) {
/*************************************************************************************
* simple encrption cipher = 1 + plain * n mod n^2
* in:
*   plains: plain text(2048 bits)
* out:
*   ciphers: encrypted result.
*/
  int tid=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(tid>=count)
    return;
  context_t      bn_context(cgbn_report_monitor, report, tid);  
  env_cph_t      bn_env(bn_context.env<env_cph_t>());                   
  env_cph_t::cgbn_t  n, nsquare, plain, cipher;
  cgbn_load(bn_env, n, &gpu_pub_key[0].n);      
  cgbn_load(bn_env, plain, plains + tid);
  cgbn_load(bn_env, nsquare, &gpu_pub_key[0].nsquare);
  cgbn_load(bn_env, plain, plains + tid);
  cgbn_mul(bn_env, cipher, n, plain);
  cgbn_add_ui32(bn_env, cipher, cipher, 1);
  cgbn_rem(bn_env, cipher, cipher, nsquare);

  cgbn_store(bn_env, ciphers + tid, cipher);   // store r into sum
}

__global__ __noinline__ void raw_encrypt_with_obfs(PaillierPublicKey *gpu_pub_key, cgbn_error_report_t *report, 
		gpu_cph *plains, gpu_cph *ciphers, int count, uint32_t *rand_vals) {
/*******************************************************************************
* encryption and obfuscation in one function, with less memory copy.
* in:
*   plains: plain text.
*   state: random generator state.
* out:
*   ciphers: encrpted text.
*/
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int tid= idx/TPI;
  if(tid>=count)
    return;
  context_t     bn_context(cgbn_report_monitor, report, tid);  
  env_cph_t     bn_env(bn_context.env<env_cph_t>());                   
  env_cph_t::cgbn_t  n, nsquare, plain,  tmp, max_int, cipher; 
  env_cph_t::cgbn_wide_t tmp_wide;
  cgbn_load(bn_env, n, &gpu_pub_key[0].n);      
  cgbn_load(bn_env, plain, plains + tid);
  cgbn_load(bn_env, nsquare, &gpu_pub_key[0].nsquare);
  cgbn_load(bn_env, max_int, &gpu_pub_key[0].max_int);
  cgbn_load(bn_env, plain, plains + tid);
  cgbn_sub(bn_env, tmp, n, max_int); 
  cgbn_mul(bn_env, cipher, n, plain);
  cgbn_add_ui32(bn_env, cipher, cipher, 1);
  cgbn_rem(bn_env, cipher, cipher, nsquare);

  env_cph_t::cgbn_t r; 

  // hiprandState localState = state[idx];
  uint32_t rand_r = rand_vals[tid];
  // state[idx] = localState;

  cgbn_set_ui32(bn_env, r, rand_r); // TODO: new rand or reuse

  mont_modular_power(bn_env,tmp, r, n, nsquare);

  cgbn_mul_wide(bn_env, tmp_wide, cipher, tmp);
  cgbn_rem_wide(bn_env, r, tmp_wide, nsquare);
  cgbn_store(bn_env, ciphers + tid, r);   // store r into sum
}


__global__ __noinline__ void raw_add(PaillierPublicKey *gpu_pub_key, cgbn_error_report_t *report, gpu_cph *ciphers_r, 
		gpu_cph *ciphers_a, gpu_cph *ciphers_b,int count) {
/**************************************************************************************
* add under encrypted text.
* in: 
*   ciphers_a, ciphers_b: encrypted a and b.
* out:
*   ciphers_r: encrypted result.
*/
  int tid=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(tid>=count)
    return;
  context_t      bn_context(cgbn_report_monitor, report, tid);  
  env_cph_t          bn_env(bn_context.env<env_cph_t>());                   
  env_cph_t::cgbn_t  nsquare, r, a, b;
  env_cph_t::cgbn_wide_t r_wide;
  cgbn_load(bn_env, nsquare, &gpu_pub_key[0].nsquare);      
  cgbn_load(bn_env, a, ciphers_a + tid);      
  cgbn_load(bn_env, b, ciphers_b + tid);
  cgbn_mul_wide(bn_env, r_wide, a, b);
  cgbn_rem_wide(bn_env, r, r_wide, nsquare);
  cgbn_store(bn_env, ciphers_r + tid, r);
}

__global__ void raw_mul(PaillierPublicKey *gpu_pub_key, cgbn_error_report_t *report, gpu_cph *ciphers_r, 
		gpu_cph *ciphers_a, gpu_cph *plains_b,int count) {
/****************************************************************************************
* multiplication under encrypted text. b * a.
* in:
*   ciphers_a, plains_b: encrypted a and b.
* out:
*   ciphers_r: encrypted result.
*/
  int tid=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(tid>=count)
    return;
  context_t      bn_context(cgbn_report_monitor, report, tid);  
  env_cph_t      bn_env(bn_context.env<env_cph_t>());                   
  env_cph_t::cgbn_t  n,max_int, nsquare, r, cipher, plain, neg_c, neg_scalar,tmp;               

  cgbn_load(bn_env, n, &gpu_pub_key[0].n);      
  cgbn_load(bn_env, max_int, &gpu_pub_key[0].max_int);      
  cgbn_load(bn_env, nsquare, &gpu_pub_key[0].nsquare);      
  cgbn_load(bn_env, cipher, ciphers_a + tid);      
  cgbn_load(bn_env, plain, plains_b + tid);

  cgbn_sub(bn_env, tmp, n, max_int); 
  if(cgbn_compare(bn_env, plain, tmp) >= 0 ) {
    // Very large plaintext, take a sneaky shortcut using inverses
    cgbn_modular_inverse(bn_env,neg_c, cipher, nsquare);
    cgbn_sub(bn_env, neg_scalar, n, plain);
    mont_modular_power(bn_env, r, neg_c, neg_scalar, nsquare);
  } else {
    mont_modular_power(bn_env, r, cipher, plain, nsquare);
  }
  cgbn_store(bn_env, ciphers_r + tid, r);
}

  
__global__ void raw_decrypt(PaillierPrivateKey *gpu_priv_key, PaillierPublicKey *gpu_pub_key,
	   	cgbn_error_report_t *report, gpu_cph *plains, gpu_cph *ciphers, int count) {
/*************************************************************************************
* decryption
* in:
*   ciphers: encrypted text. 2048 bits.
* out:
*   plains: decrypted plain text.
*/
  int tid=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(tid>=count)
    return;
  context_t      bn_context(cgbn_report_monitor, report, tid);
  env_cph_t          bn_env(bn_context.env<env_cph_t>());
  env_cph_t::cgbn_t  mp, mq, tmp, q_inverse, n, p, q, hp, hq, psquare, qsquare, cipher;  
  cgbn_load(bn_env, cipher, ciphers + tid);
  cgbn_load(bn_env, q_inverse, &gpu_priv_key[0].q_inverse);
  cgbn_load(bn_env, n, &gpu_pub_key[0].n);
  cgbn_load(bn_env, p, &gpu_priv_key[0].p);
  cgbn_load(bn_env, q, &gpu_priv_key[0].q);
  cgbn_load(bn_env, hp, &gpu_priv_key[0].hp);
  cgbn_load(bn_env, hq, &gpu_priv_key[0].hq);
  cgbn_load(bn_env, psquare, &gpu_priv_key[0].psquare);
  cgbn_load(bn_env, qsquare, &gpu_priv_key[0].qsquare);
  
  l_func(bn_env, mp, cipher, p, psquare, hp); 
  l_func(bn_env, mq, cipher, q, qsquare, hq); 
  
  cgbn_sub(bn_env, tmp, mp, mq);
  cgbn_mul(bn_env, tmp, tmp, q_inverse); 
  cgbn_rem(bn_env, tmp, tmp, p);
  cgbn_mul(bn_env, tmp, tmp, q);
  cgbn_add(bn_env, tmp, mq, tmp);
  cgbn_rem(bn_env, tmp, tmp, n);
  
  cgbn_store(bn_env, plains + tid, tmp);
}

void print_buffer_in_hex(char *addr, int count) {
  printf("dumping memory in hex\n");
  for (int i = 0; i < count; i++)
    printf("%x", *(addr + i) & 0xff); // remove padding.
  printf("\n");
}

void print_num_hex(char *addr, int count) {
  printf("dumping memory in hex, little endine\n");
  bool leading = false;
  for (int i = count - 1; i >= 0; i--) {
    if (*(addr + i) >= 0 && leading == false) {
      leading = true;
      printf("%x", *(addr + i) & 0xff); // remove padding.
    } else if (leading == true) {
      printf("%x", *(addr + i) & 0xff); // remove padding.
    }
  }
  printf("\n");
}

extern "C" {
PaillierPublicKey* gpu_pub_key;
PaillierPrivateKey* gpu_priv_key;
cgbn_error_report_t* err_report;

void init_pub_key(void *n, void *g, void *nsquare, void *max_int) {
  hipMalloc(&gpu_pub_key, sizeof(PaillierPublicKey));
  hipMemcpy((void *)&gpu_pub_key->g, g, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_pub_key->n, n, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_pub_key->nsquare, nsquare, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_pub_key->max_int, max_int, CPH_BITS/8, hipMemcpyHostToDevice);
}

void init_priv_key(void *p, void *q, void *psquare, void *qsquare, void *q_inverse,
                   void *hp, void *hq) {
  hipMalloc(&gpu_priv_key, sizeof(PaillierPrivateKey));
  hipMemcpy((void *)&gpu_priv_key->p, p, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_priv_key->q, q, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_priv_key->psquare, psquare, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_priv_key->qsquare, qsquare, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_priv_key->q_inverse, q_inverse, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_priv_key->hp, hp, CPH_BITS/8, hipMemcpyHostToDevice);
  hipMemcpy((void *)&gpu_priv_key->hq, hq, CPH_BITS/8, hipMemcpyHostToDevice);
}

void init_err_report() {
  CUDA_CHECK(cgbn_error_report_alloc(&err_report));
}

void reset() {
  CUDA_CHECK(cgbn_error_report_free(err_report));
  hipFree(gpu_pub_key);
  hipFree(gpu_priv_key);
}

void call_raw_encrypt_obfs(gpu_cph *plains_on_gpu, const uint32_t count,  \
  gpu_cph *ciphers_on_gpu, uint32_t* rand_vals_gpu) {
  // all parameters on gpu

  int TPB = 128;
  int IPB = TPB/TPI;
  int block_size = (count + IPB - 1)/IPB;
  int thread_size = TPB;
  if (rand_vals_gpu != NULL) {
    raw_encrypt_with_obfs<<<block_size, thread_size>>>(gpu_pub_key, err_report, \
      plains_on_gpu, ciphers_on_gpu, count, rand_vals_gpu);
  }
  else
    raw_encrypt<<<block_size, thread_size>>>(gpu_pub_key, err_report, plains_on_gpu,\
       ciphers_on_gpu, count);

}

void call_raw_add(gpu_cph *cipher_a, gpu_cph *cipher_b, gpu_cph *cipher_res, const uint32_t count) {
  
  int TPB = 128;
  int IPB = TPB/TPI;

  int block_size = (count + IPB - 1) / IPB;
  int thread_size = TPB;

  raw_add<<<block_size, thread_size>>>(gpu_pub_key, err_report, cipher_res, cipher_a, cipher_b, count);

}

void call_raw_mul(gpu_cph *cipher_a, plain_t *plain_b, gpu_cph *cipher_res, const uint32_t count) {
  // a is cipher, b is plain
  gpu_cph *plain_b_ext;
  
  int TPB = 128;
  int IPB = TPB/TPI;

  cudaMallocAndSet((void **)&plain_b_ext, sizeof(gpu_cph) * count);

  for (int i = 0; i < count; i++)
    hipMemcpy(plain_b_ext + i, plain_b + i, sizeof(plain_t), hipMemcpyDeviceToDevice);
  
  int block_size = (count + IPB - 1) / IPB;
  int thread_size = TPB;

  raw_mul<<<block_size, thread_size>>>(gpu_pub_key, err_report, cipher_res, cipher_a, \
     plain_b_ext, count);

  hipFree(plain_b_ext);
}

void call_raw_decrypt(gpu_cph *cipher_gpu, const uint32_t count, plain_t *res) {
  gpu_cph *plain_gpu;
  
  hipMalloc((void **)&plain_gpu, sizeof(gpu_cph) * count);
  hipMemset(plain_gpu, 0, sizeof(gpu_cph) * count);
  
  int TPB = 128;
  int IPB = TPB/TPI;
  int block_size = (count + IPB - 1) / IPB;
  int thread_size = TPB;

  raw_decrypt<<<block_size, thread_size>>>(gpu_priv_key, gpu_pub_key, err_report, plain_gpu, \
  cipher_gpu, count);

  for (int i = 0; i < count; i++)
    hipMemcpy(res + i, plain_gpu + i, sizeof(plain_t), hipMemcpyDeviceToHost);

  hipFree(plain_gpu);
}

void cipher_align(PaillierEncryptedNumber *a, PaillierEncryptedNumber *b, const uint32_t count) {
  // align exponent before executing "encrypted add" operation
  // parameters:
  //   a: PEN array on cpu, b: same as a
  // steps:
  //   1. figure out whose exponent is bigger
  //   2. update exponent
  //   3. perform raw mul
  //   4. copy back to PaillierEncryptedNumber
  int *map = (int *) malloc(sizeof(int) * count);
  plain_t *cof;
  hipMallocManaged(&cof, sizeof(plain_t) * count);
  // 1
  for (int i = 0; i < count; i++) {
    map[i] = a[i].exponent < b[i].exponent ? 0 : 1;
    cof[i] = (plain_t) pow(a[i].base, abs(a[i].exponent- b[i].exponent));
    if (a[i].exponent < b[i].exponent)
      a[i].exponent = b[i].exponent;
    else b[i].exponent = a[i].exponent;
  }
  // dumpMem(a[0].cipher, sizeof(gpu_cph));
  gpu_cph *encoding;
  gpu_cph *res;
  
  hipMalloc(&encoding, sizeof(gpu_cph) * count);
  hipMalloc(&res, sizeof(gpu_cph) * count);
  for (int i = 0; i < count; i++) {
    if (map[i] == 0)
      hipMemcpy(encoding + i, a + i, sizeof(gpu_cph), hipMemcpyHostToDevice);
    else
      hipMemcpy(encoding + i, b + i, sizeof(gpu_cph), hipMemcpyHostToDevice);
  }
  // 2
  call_raw_mul(encoding, cof, res, count);
  // 3
  for (int i = 0; i < count; i++) {
    if (map[i] == 0)
      hipMemcpy((a + i)->cipher, res + i, sizeof(gpu_cph), hipMemcpyDeviceToHost);
    else
      hipMemcpy((b + i)->cipher, res + i, sizeof(gpu_cph), hipMemcpyDeviceToHost);
  }
  
  //..
  hipFree(encoding);
  hipFree(res);
  free(map);
  hipFree(cof);
 
}

void increase_exponent_to(PaillierEncryptedNumber *a, uint32_t *inc, PaillierEncryptedNumber *res) {
  
}

void cipher_add_cipher(PaillierEncryptedNumber *a, PaillierEncryptedNumber *b, \
  PaillierEncryptedNumber *r, const uint32_t count) {
  // perform encrypted add on PEN
  // parameters:
  //   a, b: add numbers, on cpu. c: result on cpu
  // steps:
  //   1. align
  //   2. perform raw add
  //   3. copy to cpu
  cipher_align(a, b, count);
  gpu_cph *cipher_a;
  gpu_cph *cipher_b;
  gpu_cph *cipher_res;
  cudaMallocAndSet((void **)&cipher_a, sizeof(gpu_cph) * count);
  cudaMallocAndSet((void **)&cipher_b, sizeof(gpu_cph) * count);
  cudaMallocAndSet((void **)&cipher_res, sizeof(gpu_cph) * count);

  extractPen(cipher_a, a, count, HostToDevice);
  extractPen(cipher_b, b, count, HostToDevice);

  call_raw_add(cipher_a, cipher_b, cipher_res, count);
  penFromBuffer(r, cipher_res, count, DeviceToHost);

  for (int i = 0; i < count; i++) {
    r[i].exponent = a[i].exponent;
    r[i].base = a[i].base;
  }
  
}

void plain_mul_cipher(FixedPointNumber *b, PaillierEncryptedNumber *a, \
   PaillierEncryptedNumber *r, const int count) {
  // perform encrypted multiplication
  // parameters:
  //   b: coefficients, plain text on cpu
  //   a: encrypted num of arrays
  //   r: result, all on cpu
  // steps:
  //   1. perform raw mul
  //   2. add exponent together.
  //   3. copy to cpu
  plain_t *plain_gpu;
  gpu_cph *cipher_gpu;
  gpu_cph *cipher_res;
  cudaMallocAndSet((void **)&plain_gpu, sizeof(plain_t) * count);
  cudaMallocAndSet((void **)&cipher_gpu, sizeof(gpu_cph) * count);
  cudaMallocAndSet((void **)&cipher_res, sizeof(gpu_cph) * count);

  extractPen(cipher_gpu, a, count, HostToDevice);
  for (int i = 0; i < count; i++)
    hipMemcpy(plain_gpu + i, &((b + i)->encoding), sizeof(plain_t), hipMemcpyHostToDevice);
  
  call_raw_mul(cipher_gpu, plain_gpu, cipher_res, count);
  for (int i = 0; i < count; i++) {
    hipMemcpy((r + i)->cipher, cipher_res + i, sizeof(gpu_cph), hipMemcpyDeviceToHost);
    (r + i)->base = (a + i)->base;
    (r + i)->exponent = (a + i)->exponent + (b + i)->exponent;
  }

  hipFree(plain_gpu);
  hipFree(cipher_gpu);
  hipFree(cipher_res);
}

void encrypt(FixedPointNumber *plain, gpu_cph *r, const int32_t count, const bool obf) {
  // encrypt function.
  // parameters:
  //   plain: in cpu
  //   r : in cpu
  // steps:
  //   1. copy encoding to gpu
  //   2. perform raw encrypt
  //   3. copy back to result(on cpu)
  gpu_cph *raw_plain_gpu;
  gpu_cph *raw_cipher_gpu;
  unsigned int *obfs = NULL;
  hipMalloc(&raw_plain_gpu, sizeof(gpu_cph) * count);
  hipMalloc(&raw_cipher_gpu, sizeof(gpu_cph) * count);
  memset(r, 0, sizeof(gpu_cph) * count);
  hipMemset(raw_plain_gpu, 0, sizeof(gpu_cph) * count);
  
  for (int i = 0; i < count; i++) {
    hipMemcpy(raw_plain_gpu + i, &plain[i].encoding, sizeof(plain_t), hipMemcpyHostToDevice);
  }
  
  if (obf) {
    hipMallocManaged(&obfs, sizeof(unsigned int) * count);
    for (int i = 0; i < count; i++) obfs[i] = rand();
  }
   
  call_raw_encrypt_obfs(raw_plain_gpu, count, raw_cipher_gpu, obfs);

  hipMemcpy(r, raw_cipher_gpu, sizeof(gpu_cph) * count, hipMemcpyDeviceToHost);

  hipFree(raw_plain_gpu);
  hipFree(raw_cipher_gpu);
  if (obf) hipFree(obfs);
}

void decrypt(PaillierEncryptedNumber *cipher, plain_t *r, const int32_t count) {
  // perform decrypt
  // parameters:
  //   cipher: in cpu
  //   r : in cpu
  // steps:
  //   1. copy to gpu
  //   2. perform raw decrypt
  //   3. copy back to cpu
  gpu_cph *raw_cipher_gpu;
  hipMalloc(&raw_cipher_gpu, sizeof(gpu_cph) * count);
  memset(r, 0, sizeof(plain_t) * count);

  // dumpMem(cipher[0].cipher, sizeof(gpu_cph));

  for (int i = 0; i < count; i++)
    hipMemcpy(raw_cipher_gpu + i, cipher[i].cipher, sizeof(gpu_cph), hipMemcpyHostToDevice);
  
  call_raw_decrypt(raw_cipher_gpu, count, r);

  hipFree(raw_cipher_gpu);
}


void sum(PaillierEncryptedNumber *cipher, PaillierEncryptedNumber *res, const int32_t count) {
  // sum
  // parameters:
  //  cipher: in cpu
  //  r: in cpu
  // steps:
  //  1. copy to gpu
  //  2. align
  //  3. perform raw add on half
  //  4. loop until only one left

  // if count is odd, add one
  printf("count: %d\n", count);
  int32_t num_elem = count % 2 == 1 ? count + 1 : count;
  gpu_cph *ciphers_buf[2];
  plain_t *inc;
  cudaMallocAndSet((void **)&ciphers_buf[0], sizeof(gpu_cph) * num_elem);
  cudaMallocAndSet((void **)&ciphers_buf[1], sizeof(gpu_cph) * num_elem);
  hipMallocManaged((void **)&inc, sizeof(plain_t) * count);

  uint32_t max_exponent = 0;
  for (int i = 0; i < count; i++)
    max_exponent = max_exponent < cipher[i].exponent ? cipher[i].exponent : max_exponent;
  for (int i = 0; i < count; i++) {
    inc[i] = (int32_t) pow(cipher[i].base, max_exponent - cipher[i].exponent);
    printf("inc[%d]: %d\n", i, inc[i]);
  }

  extractPen(ciphers_buf[0], cipher, count, HostToDevice);
  call_raw_mul(ciphers_buf[0], inc, ciphers_buf[1], count);
  
  if (count % 2 == 1)
    hipMemset(ciphers_buf[1] + num_elem - 1, 1, 1);
    
  uint32_t dst_index = 0;
  gpu_cph *dst_buf;
  gpu_cph *src_buf;
  for (int i = num_elem / 2; i >= 1; i /= 2) {
    dst_buf = ciphers_buf[dst_index % 2];
    src_buf = ciphers_buf[(dst_index % 2 + 1) % 2];
    printf("check it %d\n", i);
    call_raw_add(src_buf, src_buf + i, dst_buf, i);
    dst_index += 1;
  }

  hipMemcpy(res->cipher, dst_buf, sizeof(gpu_cph), hipMemcpyDeviceToHost);
  res->base = cipher[0].base;
  res->exponent = max_exponent;

  hipFree(ciphers_buf[0]);
  hipFree(ciphers_buf[1]);
  hipFree(inc);
}

}// extern "C"